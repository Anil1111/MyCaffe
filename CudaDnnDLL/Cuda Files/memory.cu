#include "hip/hip_runtime.h"
//=============================================================================
//	FILE:	memory.cu
//
//	DESC:	This file the basic memory management for the given device
//=============================================================================

#include "memory.h"

//=============================================================================
//	Class Methods
//=============================================================================

template <class T>
Memory<T>::Memory() : m_memory(), m_memoryPointers(), m_hostbuffers(), m_streams(), m_tensorDesc(), m_filterDesc(), m_convDesc(), m_poolDesc(), m_lrnDesc(), m_cudnn(), m_pca(), m_tsnegp(), m_tsneg(), m_memtest(), m_nccl()
{
	m_memory.SetMemoryPointers(&m_memoryPointers);

	m_tOne = (T)1;
	m_tZero = (T)0;

#ifdef CUDNN_5
	CreateActivationDesc(&m_hGlobalActivationSigmoid);
	SetActivationDesc(m_hGlobalActivationSigmoid, SIGMOID);
	CreateActivationDesc(&m_hGlobalActivationRelu);
	SetActivationDesc(m_hGlobalActivationRelu, RELU);
	CreateActivationDesc(&m_hGlobalActivationTanh);
	SetActivationDesc(m_hGlobalActivationTanh, TANH);
#endif
}

template Memory<double>::Memory();
template Memory<float>::Memory();


template <class T>
Memory<T>::~Memory()
{
	for (int i=0; i<m_hostbuffers.GetCount(); i++)
	{
		FreeHostBuffer(i);
	}

	for (int i=0; i<m_streams.GetCount(); i++)
	{
		FreeStream(i);
	}

	for (int i=0; i<m_tensorDesc.GetCount(); i++)
	{
		FreeTensorDesc(i);
	}

	for (int i=0; i<m_filterDesc.GetCount(); i++)
	{
		FreeFilterDesc(i);
	}

	for (int i=0; i<m_convDesc.GetCount(); i++)
	{
		FreeConvolutionDesc(i);
	}

	for (int i=0; i<m_poolDesc.GetCount(); i++)
	{
		FreePoolingDesc(i);
	}

	for (int i=0; i<m_lrnDesc.GetCount(); i++)
	{
		FreeLRNDesc(i);
	}

	for (int i=0; i<m_cudnn.GetCount(); i++)
	{
		FreeCuDNN(i);
	}

#ifdef CUDNN_5
	for (int i=0; i<m_activationDesc.GetCount(); i++)
	{
		FreeActivationDesc(i);
	}

	m_hGlobalActivationSigmoid = 0;
	m_hGlobalActivationRelu = 0;
	m_hGlobalActivationTanh = 0;

	for (int i = 0; i < m_dropoutDesc.GetCount(); i++)
	{
		FreeDropoutDesc(i);
	}
#endif

	for (int i=0; i<m_pca.GetCount(); i++)
	{
		FreePCA(i);
	}

	for (int i=0; i<m_tsnegp.GetCount(); i++)
	{
		FreeTsneGaussianPerplexity(i);
	}

	for (int i = 0; i < m_memtest.GetCount(); i++)
	{
		FreeMemoryTest(i);
	}

	for (int i = 0; i < m_nccl.GetCount(); i++)
	{
		FreeNCCL(i);
	}
}

template Memory<double>::~Memory();
template Memory<float>::~Memory();


template <class T>
long Memory<T>::GetDeviceMemory(int nDeviceID, T* pfTotal, T* pfFree, T* pfUsed, bool* pbEstimate)
{
	LONG lErr;
	size_t lFree = 0;
	size_t lTotal = 0;
	size_t lUsed = 0;
	int nOriginalDeviceID = -1;

	if (nDeviceID >= 0)
	{
		if (lErr = hipGetDevice(&nOriginalDeviceID))
			return lErr;

		if (nDeviceID != nOriginalDeviceID)
		{
			if (lErr = hipSetDevice(nDeviceID))
				return lErr;
		}
	}

	if (nDeviceID == -1)
	{
		hipDeviceProp_t prop;

		memset(&prop, 0, sizeof(hipDeviceProp_t));
		if (lErr = hipGetDeviceProperties(&prop, nDeviceID))
			return lErr;

		lTotal = prop.totalGlobalMem;
		lUsed = (size_t)m_memory.GetTotalUsed();
		lFree = lTotal - lUsed;
		*pbEstimate = true;
	}
	else
	{
		if (lErr = hipMemGetInfo(&lFree, &lTotal))
			return lErr;

		lUsed = lTotal - lFree;
		*pbEstimate = false;
	}

	*pfTotal = (T)((double)lTotal / (double)1000000000.0);
	*pfFree = (T)((double)lFree / (double)1000000000.0);
	*pfUsed = (T)((double)lUsed / (double)1000000000.0);

	if (nOriginalDeviceID >= 0 && nOriginalDeviceID != nDeviceID)
	{
		if (lErr = hipSetDevice(nOriginalDeviceID))
			return lErr;
	}

	return 0;
}

template long Memory<double>::GetDeviceMemory(int nDeviceID, double* pdfTotal, double* pdfFree, double* pdfUsed, bool* pbEstimate);
template long Memory<float>::GetDeviceMemory(int nDeviceID, float* pfTotal, float* pfFree, float* pfUsed, bool* pbEstimate);


template <class T>
long Memory<T>::AllocHost(LPTSTR* ppDst, LPTSTR pSrc)
{
	int nLen = (int)_tcslen(pSrc);

	if (nLen == 0)
		return ERROR_PARAM_OUT_OF_RANGE;

	nLen++;	// make room for NULL;

	LPTSTR pDst = NULL;
	LONG lSize = nLen * sizeof(TCHAR);
	LONG lErr = 0;

#ifdef USE_PINNED_HOST_MEM
	if (lErr = hipHostMalloc(&pDst, lSize))
		return lErr;
#else
	pDst = (LPTSTR)malloc(lSize);
	if (pDst == NULL)
		return ERROR_MEMORY_OUT;
#endif

	pDst[nLen] = (TCHAR)NULL;
	_tcsncpy(pDst, pSrc, nLen);

	*ppDst = pDst;

	return lErr;
}

template long Memory<double>::AllocHost(LPTSTR* ppDst, LPTSTR pSrc);
template long Memory<float>::AllocHost(LPTSTR* ppDst, LPTSTR pSrc);


template <class T>
long Memory<T>::AllocHost(long lCount, T** ppDst, T* pSrc, bool bSrcOnDevice)
{
	if (lCount == 0)
		return ERROR_PARAM_OUT_OF_RANGE;

	if (ppDst == NULL)
		return ERROR_PARAM_NULL;

	long lSize = lCount * sizeof(T);
	T* pDst = NULL;	
	LONG lErr = 0;

#ifdef USE_PINNED_HOST_MEM
	if (lErr = hipHostMalloc(&pDst, lSize))
		return lErr;
#else
	pDst = (T*)malloc(lSize);
	if (pDst == NULL)
		return ERROR_MEMORY_OUT;
#endif

	if (pSrc != NULL)
	{
		hipMemcpyKind kind = (bSrcOnDevice) ? hipMemcpyDeviceToHost : hipMemcpyHostToHost;

		if (lErr = hipMemcpy(pDst, pSrc, lSize, kind))
		{
#ifdef USE_PINNED_HOST_MEM
			hipHostFree(pDst);
#else
			free(pDst);
#endif
			return lErr;
		}
	}
	else
	{
		memset(pDst, 0, lSize);
	}

	*ppDst = pDst;
	return hipGetLastError();
}

template long Memory<double>::AllocHost(long lCount, double** ppDst, double* pSrc, bool bSrcOnDevice);
template long Memory<float>::AllocHost(long lCount, float** ppDst, float* pSrc, bool bSrcOnDevice);


template <class T>
long Memory<T>::CopyToHost(long lCount, T* pDst, T* pSrc, bool bSrcOnDevice)
{
	if (lCount == 0)
		return ERROR_PARAM_OUT_OF_RANGE;

	if (pDst == NULL || pSrc == NULL)
		return ERROR_PARAM_NULL;

	hipMemcpyKind kind = (bSrcOnDevice) ? hipMemcpyDeviceToHost : hipMemcpyHostToHost;

	return hipMemcpy(pDst, pSrc, lCount * sizeof(T), kind);
}

template long Memory<double>::CopyToHost(long lCount, double* pDst, double* pSrc, bool bSrcOnDevice);
template long Memory<float>::CopyToHost(long lCount, float* pDst, float* pSrc, bool bSrcOnDevice);


template <class T>
long Memory<T>::AllocHostBuffer(long lCount, long* phHandle)
{
	LONG lErr = 0;

	if (lCount % 2 != 0)
		lCount++;

	T* pMem = NULL;
	
	if (lErr = AllocHost(lCount, &pMem, NULL, FALSE))
		return lErr;

	HostBuffer<T>* pHostBuf = new HostBuffer<T>(pMem, lCount);
	if (pHostBuf == NULL)
	{
		FreeHost(pMem);
		return ERROR_MEMORY_OUT;
	}

	long hHandle = m_hostbuffers.Allocate(pHostBuf);
	if (hHandle < 0)
	{
		delete pHostBuf;
		FreeHost(pMem);
		return ERROR_MEMORY_OUT;
	}

	m_rgActiveHostBuffers.push_back(pHostBuf);

	*phHandle = hHandle;

	return 0;
}

template long Memory<double>::AllocHostBuffer(long lCount, long* phHandle);
template long Memory<float>::AllocHostBuffer(long lCount, long* phHandle);


template <class T>
long Memory<T>::FreeHostBuffer(long hHandle)
{
	HostBuffer<T>* pHostBuf = (HostBuffer<T>*)m_hostbuffers.Free(hHandle);
	
	if (pHostBuf != NULL)
	{
		if (pHostBuf->Data() != NULL)
			FreeHost(pHostBuf->Data());

		std::remove(m_rgActiveHostBuffers.begin(), m_rgActiveHostBuffers.end(), pHostBuf);

		delete pHostBuf;
	}

	return 0;
}

template long Memory<double>::FreeHostBuffer(long hHandle);
template long Memory<float>::FreeHostBuffer(long hHandle);


template <class T>
bool Memory<T>::IsHostBuffer(T* pf)
{
	int nCount = (int)m_rgActiveHostBuffers.size();

	for (int i=0; i<nCount; i++)
	{
		if (m_rgActiveHostBuffers[i]->Data() == pf)
			return true;
	}

	return false;
}

template bool Memory<double>::IsHostBuffer(double* pf);
template bool Memory<float>::IsHostBuffer(float* pf);


template <class T>
long Memory<T>::CreateStream(long* phHandle, bool bNonBlocking)
{
	LONG lErr;
	hipStream_t stream = NULL;

	if (phHandle == NULL)
		return ERROR_PARAM_NULL;

	if (bNonBlocking)
	{
		if (lErr = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking))
			return lErr;
	}
	else
	{
		if (lErr = hipStreamCreate(&stream))
			return lErr;
	}

	long hHandle = m_streams.Allocate(stream);
	if (hHandle < 0)
	{
		hipStreamDestroy(stream);
		return ERROR_MEMORY_OUT;
	}

	*phHandle = hHandle;
	return 0;
}

template long Memory<double>::CreateStream(long* phHandle, bool bNonBlocking);
template long Memory<float>::CreateStream(long* phHandle, bool bNonBlocking);


template <typename T>
__global__ void synchronize_thread_kernel()
{
}

template <class T>
long Memory<T>::SynchronizeThread()
{
	synchronize_thread_kernel<T><<<1, 1>>>();
	return hipGetLastError();
}

template long Memory<double>::SynchronizeThread();
template long Memory<float>::SynchronizeThread();



template <class T>
long Memory<T>::CreateCuDNN(long hStream, long* phHandle)
{
	LONG lErr;
	hipdnnHandle_t cudnn = NULL;

	if (phHandle == NULL)
		return ERROR_PARAM_NULL;

	if (lErr = hipdnnCreate(&cudnn))
		return lErr | ERROR_CUDNN_OFFSET;

	if (hStream > 0)
	{
		if (lErr = hipdnnSetStream(cudnn, GetStream(hStream)))
			return lErr | ERROR_CUDNN_OFFSET;
	}

	long hHandle = m_cudnn.Allocate(cudnn);
	if (hHandle < 0)
	{
		hipdnnDestroy(cudnn);
		return ERROR_MEMORY_OUT;
	}

	*phHandle = hHandle;
	return 0;
}

template long Memory<double>::CreateCuDNN(long hStream, long* phHandle);
template long Memory<float>::CreateCuDNN(long hStream, long* phHandle);


template <class T>
long Memory<T>::CreateTensorDesc(long* phHandle)
{
	LONG lErr;
	hipdnnTensorDescriptor_t desc = NULL;

	if (phHandle == NULL)
		return ERROR_PARAM_NULL;

	if (lErr = hipdnnCreateTensorDescriptor(&desc))
		return lErr | ERROR_CUDNN_OFFSET;

	long hHandle = m_tensorDesc.Allocate(desc);
	if (hHandle < 0)
	{
		hipdnnDestroyTensorDescriptor(desc);
		return ERROR_MEMORY_OUT;
	}

	*phHandle = hHandle;
	return 0;
}

template long Memory<double>::CreateTensorDesc(long* phHandle);
template long Memory<float>::CreateTensorDesc(long* phHandle);


template <class T>
long Memory<T>::AddTensor(long hHandle, T fAlpha, long hSrcDesc, long hSrc, int nSrcOffset, T fBeta, long hDstDesc, long hDst, int nDstOffset)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnTensorDescriptor_t srcdesc = GetTensorDesc(hSrcDesc);
	hipdnnTensorDescriptor_t dstdesc = GetTensorDesc(hDstDesc);
	MemoryItem* pSrc;
	MemoryItem* pDst;

	if (lErr = m_memory.GetData(hSrc, &pSrc))
		return lErr;

	if (lErr = m_memory.GetData(hDst, &pDst))
		return lErr;

	if (cudnn == NULL || srcdesc == NULL || dstdesc == NULL)
		return ERROR_PARAM_NULL;

	T* src = (T*)pSrc->Data();
	T* dst = (T*)pDst->Data();

	if (nSrcOffset > 0)
		src += nSrcOffset;

	if (nDstOffset > 0)
		dst += nDstOffset;

#ifdef CUDNN_4
	if (lErr = hipdnnAddTensor(cudnn, &fAlpha, srcdesc, src, &fBeta, dstdesc, dst))
		return lErr | ERROR_CUDNN_OFFSET;
#else
	if (lErr = hipdnnAddTensor(cudnn, CUDNN_ADD_SAME_C, &fAlpha, srcdesc, src, &fBeta, dstdesc, dst))
		return lErr | ERROR_CUDNN_OFFSET;
#endif
	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::AddTensor(long hHandle, double dfAlpha, long hSrcDesc, long hSrc, int nSrcOffset, double dfBeta, long hDstDesc, long hDst, int nDstOffset);
template long Memory<float>::AddTensor(long hHandle, float fAlpha, long hSrcDesc, long hSrc, int nSrcOffset, float fBeta, long hDstDesc, long hDst, int nDstOffset);


template <class T>
long Memory<T>::CreateFilterDesc(long* phHandle)
{
	LONG lErr;
	hipdnnFilterDescriptor_t desc = NULL;

	if (phHandle == NULL)
		return ERROR_PARAM_NULL;

	if (lErr = hipdnnCreateFilterDescriptor(&desc))
		return lErr | ERROR_CUDNN_OFFSET;

	long hHandle = m_filterDesc.Allocate(desc);
	if (hHandle < 0)
	{
		hipdnnDestroyFilterDescriptor(desc);
		return ERROR_MEMORY_OUT;
	}

	*phHandle = hHandle;
	return 0;
}

template long Memory<double>::CreateFilterDesc(long* phHandle);
template long Memory<float>::CreateFilterDesc(long* phHandle);


template <class T>
long Memory<T>::CreateConvolutionDesc(long* phHandle)
{
	LONG lErr;
	hipdnnConvolutionDescriptor_t desc = NULL;

	if (phHandle == NULL)
		return ERROR_PARAM_NULL;

	if (lErr = hipdnnCreateConvolutionDescriptor(&desc))
		return lErr | ERROR_CUDNN_OFFSET;

	long hHandle = m_convDesc.Allocate(desc);
	if (hHandle < 0)
	{
		hipdnnDestroyConvolutionDescriptor(desc);
		return ERROR_MEMORY_OUT;
	}

	*phHandle = hHandle;
	return 0;
}

template long Memory<double>::CreateConvolutionDesc(long* phHandle);
template long Memory<float>::CreateConvolutionDesc(long* phHandle);


template <class T>
long Memory<T>::GetConvolutionInfo(long hHandle, long hBottomDesc, long hFilterDesc, long hConvDesc, long hTopDesc, long lWsLimitInBytes, long* palgoFwd, long* plWsSizeFwd, long* palgoBwdFilter, long* plWsSizeBwdFilter, long* palgoBwdData, long* plWsSizeBwdData, int nPreferredFwdAlgo)
{
	hipdnnStatus_t lErr;	
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnTensorDescriptor_t bottom = GetTensorDesc(hBottomDesc);
	hipdnnFilterDescriptor_t filter = GetFilterDesc(hFilterDesc);
	hipdnnConvolutionDescriptor_t conv = GetConvolutionDesc(hConvDesc);
	hipdnnTensorDescriptor_t top = GetTensorDesc(hTopDesc);


	// Setup the algorithm preference.
	hipdnnConvolutionFwdPreference_t fwdPref = HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT;
	hipdnnConvolutionBwdFilterPreference_t bwdFltPref = HIPDNN_CONVOLUTION_BWD_FILTER_SPECIFY_WORKSPACE_LIMIT;
	hipdnnConvolutionBwdDataPreference_t bwdDataPref = HIPDNN_CONVOLUTION_BWD_DATA_SPECIFY_WORKSPACE_LIMIT;

	if (lWsLimitInBytes < 0)
	{
		lWsLimitInBytes = 0;
		fwdPref = HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST;
		bwdFltPref = HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST;
		bwdDataPref = HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST;
	}
	else if (lWsLimitInBytes == 0)
	{
		lWsLimitInBytes = 0;
		fwdPref = HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE;
		bwdFltPref = HIPDNN_CONVOLUTION_BWD_FILTER_NO_WORKSPACE;
		bwdDataPref = HIPDNN_CONVOLUTION_BWD_DATA_NO_WORKSPACE;
	}

	// Choose forward algorithm for convolution.
	hipdnnConvolutionFwdAlgo_t algoFwd;
	if (lErr = hipdnnGetConvolutionForwardAlgorithm(cudnn, bottom, filter, conv, top, fwdPref, lWsLimitInBytes, &algoFwd))
		return lErr | ERROR_CUDNN_OFFSET;

	// Get workspace size for forward algorithm.
	size_t szFwd = 0;
	if (lErr = hipdnnGetConvolutionForwardWorkspaceSize(cudnn, bottom, filter, conv, top, algoFwd, &szFwd))
		return lErr | ERROR_CUDNN_OFFSET;

	// HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD has been found by the native Caffe team to work better than 
	// HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM on deconvolution (which acts a bit buggy in this
	// situation.  For this reason, when using cuDnn deconvolution, the C# side sets the preferred
	// fwd algo to HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD which is used only when the workspace is less
	// than or equat to the default workspace size and no errors occur when attempting to get the
	// workspace size for WINOGRAD.  By default, the nPrefferredFwdAlgo paraeter is ignored.
	if (nPreferredFwdAlgo >= 0 && 
		algoFwd == HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM && 
		(int)algoFwd != nPreferredFwdAlgo)
	{
		size_t lWinogradWorkspaceSize = 0;
		lErr = hipdnnGetConvolutionForwardWorkspaceSize(cudnn, bottom, filter, conv, top, (hipdnnConvolutionFwdAlgo_t)nPreferredFwdAlgo, &lWinogradWorkspaceSize);
		if (lErr == HIPDNN_STATUS_SUCCESS)
		{
			if (lWinogradWorkspaceSize <= szFwd)
			{
				algoFwd = (hipdnnConvolutionFwdAlgo_t)nPreferredFwdAlgo;
				szFwd = lWinogradWorkspaceSize;
			}
		}
	}

	// Choose backward filter algorithm.
	hipdnnConvolutionBwdFilterAlgo_t algoBwdFilter;
	if (lErr = hipdnnGetConvolutionBackwardFilterAlgorithm(cudnn, bottom, top, conv, filter, bwdFltPref, lWsLimitInBytes, &algoBwdFilter))
		return lErr | ERROR_CUDNN_OFFSET;

	// Get workspace size for backward filter algorithm.
	size_t szBwdFilter = 0;
	if (lErr = hipdnnGetConvolutionBackwardFilterWorkspaceSize(cudnn, bottom, top, conv, filter, algoBwdFilter, &szBwdFilter))
		return lErr | ERROR_CUDNN_OFFSET;

	// Choose backward data algorithm.
	hipdnnConvolutionBwdDataAlgo_t algoBwdData;
	if (lErr = hipdnnGetConvolutionBackwardDataAlgorithm(cudnn, filter, top, conv, bottom, bwdDataPref, lWsLimitInBytes, &algoBwdData))
		return lErr | ERROR_CUDNN_OFFSET;

	// Get workspace size for backward data algorithm.
	size_t szBwdData = 0;
	if (lErr = hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnn, filter, top, conv, bottom, algoBwdData, &szBwdData))
		return lErr | ERROR_CUDNN_OFFSET;

	*palgoFwd = (long)algoFwd;
	*plWsSizeFwd = (long)szFwd;
	*palgoBwdFilter = (long)algoBwdFilter;
	*plWsSizeBwdFilter = (long)szBwdFilter;
	*palgoBwdData = (long)algoBwdData;
	*plWsSizeBwdData = (long)szBwdData;

	return hipSuccess;
}

template long Memory<double>::GetConvolutionInfo(long hHandle, long hBottomDesc, long hFilterDesc, long hConvDesc, long hTopDesc, long lWsLimitInBytes, long* palgoFwd, long* plWsSizeFwd, long* palgoBwdFilter, long* plWsSizeBwdFilter, long* palgoBwdData, long* plWsSizeBwdData, int nPreferredFwdAlgo);
template long Memory<float>::GetConvolutionInfo(long hHandle, long hBottomDesc, long hFilterDesc, long hConvDesc, long hTopDesc, long lWsLimitInBytes, long* palgoFwd, long* plWsSizeFwd, long* palgoBwdFilter, long* plWsSizeBwdFilter, long* palgoBwdData, long* plWsSizeBwdData, int nPreferredFwdAlgo);


template <class T>
long Memory<T>::ConvolutionForward(long hHandle, T fAlpha, long hBottomDesc, long hBottomData, int nBottomOffset, long hFilterDesc, long hWeight, int nWeightOffset, long hConvDesc, long algo, long hWorkspace, int nWorkspaceOffset, long lWorkspaceSize, T fBeta, long hTopDesc, long hTopData, int nTopOffset)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnTensorDescriptor_t btmdesc = GetTensorDesc(hBottomDesc);
	hipdnnFilterDescriptor_t filterdesc = GetFilterDesc(hFilterDesc);
	hipdnnConvolutionDescriptor_t convdesc = GetConvolutionDesc(hConvDesc);
	hipdnnTensorDescriptor_t topdesc = GetTensorDesc(hTopDesc);
	MemoryItem* pBtmData;
	MemoryItem* pTopData;
	MemoryItem* pWeight;
	MemoryItem* pWorkspace = NULL;

	if (lErr = m_memory.GetData(hBottomData, &pBtmData))
		return lErr;

	if (lErr = m_memory.GetData(hTopData, &pTopData))
		return lErr;

	if (lErr = m_memory.GetData(hWeight, &pWeight))
		return lErr;

	T* btmdata = (T*)pBtmData->Data();
	T* topdata = (T*)pTopData->Data();
	T* weight = (T*)pWeight->Data();
	T* wksp = NULL;

	if (hWorkspace != 0)
	{
		if (lErr = m_memory.GetData(hWorkspace, &pWorkspace))
			return lErr;

		wksp = (T*)pWorkspace->Data();
	}
	else if (lWorkspaceSize != 0)
	{
		return ERROR_PARAM_OUT_OF_RANGE;
	}

	if (nBottomOffset > 0)
		btmdata += nBottomOffset;

	if (nTopOffset > 0)
		topdata += nTopOffset;

	if (nWeightOffset > 0)
		weight += nWeightOffset;

	if (wksp != NULL && nWorkspaceOffset > 0)
		wksp += nWorkspaceOffset;

	if (lErr = hipdnnConvolutionForward(cudnn, &fAlpha, btmdesc, btmdata, filterdesc, weight, convdesc, (hipdnnConvolutionFwdAlgo_t)algo, wksp, lWorkspaceSize, &fBeta, topdesc, topdata))
		return lErr | ERROR_CUDNN_OFFSET;

	return hipDeviceSynchronize();
}

template long Memory<double>::ConvolutionForward(long hHandle, double dfAlpha, long hBottomDesc, long hBottomData, int nBottomOffset, long hFilterDesc, long hWeight, int nWeightOffset, long hConvDesc, long algo, long hWorkspace, int nWorkspaceOffset, long lWorkspaceSize, double dfBeta, long hTopDesc, long hTopData, int nTopOffset);
template long Memory<float>::ConvolutionForward(long hHandle, float fAlpha, long hBottomDesc, long hBottomData, int nBottomOffset, long hFilterDesc, long hWeight, int nWeightOffset, long hConvDesc, long algo, long hWorkspace, int nWorkspaceOffset, long lWorkspaceSize, float fBeta, long hTopDesc, long hTopData, int nTopOffset);


template <class T>
long Memory<T>::ConvolutionBackwardBias(long hHandle, T fAlpha, long hTopDesc, long hTopDiff, int nTopOffset, T fBeta, long hBiasDesc, long hBiasDiff, int nBiasOffset)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnTensorDescriptor_t topdesc = GetTensorDesc(hTopDesc);
	hipdnnTensorDescriptor_t biasdesc = GetTensorDesc(hBiasDesc);
	MemoryItem* pTopDiff;
	MemoryItem* pBiasDiff;

	if (lErr = m_memory.GetData(hTopDiff, &pTopDiff))
		return lErr;

	if (lErr = m_memory.GetData(hBiasDiff, &pBiasDiff))
		return lErr;

	T* topdiff = (T*)pTopDiff->Data();
	T* biasdiff = (T*)pBiasDiff->Data();

	if (nTopOffset > 0)
		topdiff += nTopOffset;

	if (nBiasOffset > 0)
		biasdiff += nBiasOffset;

	if (lErr = hipdnnConvolutionBackwardBias(cudnn, &fAlpha, topdesc, topdiff, &fBeta, biasdesc, biasdiff))
		return lErr | ERROR_CUDNN_OFFSET;

	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::ConvolutionBackwardBias(long hHandle, double dfAlpha, long hTopDesc, long hTopDiff, int nTopOffset, double dfBeta, long hBiasDesc, long hBiasDiff, int nBiasOffset);
template long Memory<float>::ConvolutionBackwardBias(long hHandle, float fAlpha, long hTopDesc, long hTopDiff, int nTopOffset, float fBeta, long hBiasDesc, long hBiasDiff, int nBiasOffset);


template <class T>
long Memory<T>::ConvolutionBackwardFilter(long hHandle, T fAlpha, long hBottomDesc, long hBottomData, int nBottomOffset, long hTopDesc, long hTopDiff, int nTopOffset, long hConvDesc, long algo, long hWorkspace, int nWorkspaceOffset, long lWorkspaceSize, T fBeta, long hFilterDesc, long hWeightDiff, int nWeightOffset)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnTensorDescriptor_t btmdesc = GetTensorDesc(hBottomDesc);
	hipdnnTensorDescriptor_t topdesc = GetTensorDesc(hTopDesc);
	hipdnnConvolutionDescriptor_t convdesc = GetConvolutionDesc(hConvDesc);
	hipdnnFilterDescriptor_t filterdesc = GetFilterDesc(hFilterDesc);
	MemoryItem* pBtmData;
	MemoryItem* pTopDiff;
	MemoryItem* pWeightDiff;
	MemoryItem* pWorkspace = NULL;

	if (lErr = m_memory.GetData(hBottomData, &pBtmData))
		return lErr;

	if (lErr = m_memory.GetData(hTopDiff, &pTopDiff))
		return lErr;

	if (lErr = m_memory.GetData(hWeightDiff, &pWeightDiff))
		return lErr;

	T* btmdata = (T*)pBtmData->Data();
	T* topdiff = (T*)pTopDiff->Data();
	T* weightdiff = (T*)pWeightDiff->Data();
	T* wksp = NULL;
	
	if (hWorkspace != 0)
	{
		if (lErr = m_memory.GetData(hWorkspace, &pWorkspace))
			return lErr;

		wksp = (T*)pWorkspace->Data();
	}
	else if (lWorkspaceSize != 0)
	{
		return ERROR_PARAM_OUT_OF_RANGE;
	}

	if (nBottomOffset > 0)
		btmdata += nBottomOffset;

	if (nTopOffset > 0)
		topdiff += nTopOffset;

	if (nWeightOffset > 0)
		weightdiff += nWeightOffset;

	if (wksp != NULL && nWorkspaceOffset > 0)
		wksp += nWorkspaceOffset;
	
#ifdef CUDNN_5
	if (lErr = hipdnnConvolutionBackwardFilter(cudnn, &fAlpha, btmdesc, btmdata, topdesc, topdiff, convdesc, (hipdnnConvolutionBwdFilterAlgo_t)algo, wksp, lWorkspaceSize, &fBeta, filterdesc, weightdiff))
		return lErr | ERROR_CUDNN_OFFSET;
#else
	if (lErr = cudnnConvolutionBackwardFilter_v3(cudnn, &fAlpha, btmdesc, btmdata, topdesc, topdiff, convdesc, (hipdnnConvolutionBwdFilterAlgo_t)algo, wksp, lWorkspaceSize, &fBeta, filterdesc, weightdiff))
		return lErr | ERROR_CUDNN_OFFSET;
#endif

	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::ConvolutionBackwardFilter(long hHandle, double dfAlpha, long hBottomDesc, long hBottomData, int nBottomOffset, long hTopDesc, long hTopDiff, int nTopOffset, long hConvDesc, long algo, long hWorkspace, int nWorkspaceOffset, long lWorkspaceSize, double dfBeta, long hFilterDesc, long hWeightDiff, int nWeightOffset);
template long Memory<float>::ConvolutionBackwardFilter(long hHandle, float fAlpha, long hBottomDesc, long hBottomData, int nBottomOffset, long hTopDesc, long hTopDiff, int nTopOffset, long hConvDesc, long algo, long hWorkspace, int nWorkspaceOffset, long lWorkspaceSize, float fBeta, long hFilterDesc, long hWeightDiff, int nWeightOffset);


template <class T>
long Memory<T>::ConvolutionBackwardData(long hHandle, T fAlpha, long hFilterDesc, long hWeight, int nWeightOffset, long hTopDesc, long hTopDiff, int nTopOffset, long hConvDesc, long algo, long hWorkspace, int nWorkspaceOffset, long lWorkspaceSize, T fBeta, long hBottomDesc, long hBottomDiff, int nBottomOffset)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnFilterDescriptor_t filterdesc = GetFilterDesc(hFilterDesc);
	hipdnnTensorDescriptor_t topdesc = GetTensorDesc(hTopDesc);
	hipdnnConvolutionDescriptor_t convdesc = GetConvolutionDesc(hConvDesc);
	hipdnnTensorDescriptor_t btmdesc = GetTensorDesc(hBottomDesc);
	MemoryItem* pWeight;
	MemoryItem* pTopDiff;
	MemoryItem* pBtmDiff;
	MemoryItem* pWorkspace = NULL;

	if (lErr = m_memory.GetData(hWeight, &pWeight))
		return lErr;

	if (lErr = m_memory.GetData(hTopDiff, &pTopDiff))
		return lErr;

	if (lErr = m_memory.GetData(hBottomDiff, &pBtmDiff))
		return lErr;

	T* weight = (T*)pWeight->Data();
	T* topdiff = (T*)pTopDiff->Data();
	T* btmdiff = (T*)pBtmDiff->Data();
	T* wksp = NULL;

	if (hWorkspace != 0)
	{
		if (lErr = m_memory.GetData(hWorkspace, &pWorkspace))
			return lErr;

		wksp = (T*)pWorkspace->Data();
	}
	else if (lWorkspaceSize != 0)
	{
		return ERROR_PARAM_OUT_OF_RANGE;
	}

	if (nWeightOffset > 0)
		weight += nWeightOffset;

	if (nTopOffset > 0)
		topdiff += nTopOffset;

	if (nBottomOffset > 0)
		btmdiff += nBottomOffset;

	if (wksp != NULL && nWorkspaceOffset > 0)
		wksp += nWorkspaceOffset;

#ifdef CUDNN_5
	if (lErr = hipdnnConvolutionBackwardData(cudnn, &fAlpha, filterdesc, weight, topdesc, topdiff, convdesc, (hipdnnConvolutionBwdDataAlgo_t)algo, wksp, lWorkspaceSize, &fBeta, btmdesc, btmdiff))
		return lErr | ERROR_CUDNN_OFFSET;
#else
	if (lErr = cudnnConvolutionBackwardData_v3(cudnn, &fAlpha, filterdesc, weight, topdesc, topdiff, convdesc, (hipdnnConvolutionBwdDataAlgo_t)algo, wksp, lWorkspaceSize, &fBeta, btmdesc, btmdiff))
		return lErr | ERROR_CUDNN_OFFSET;
#endif

	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::ConvolutionBackwardData(long hHandle, double dfAlpha, long hBottomDesc, long hBottomData, int nBottomOffset, long hTopDesc, long hTopDiff, int nTopOffset, long hConvDesc, long algo, long hWorkspace, int nWorkspaceOffset, long lWorkspaceSize, double dfBeta, long hFilterDesc, long hWeightDiff, int nWeightOffset);
template long Memory<float>::ConvolutionBackwardData(long hHandle, float fAlpha, long hBottomDesc, long hBottomData, int nBottomOffset, long hTopDesc, long hTopDiff, int nTopOffset, long hConvDesc, long algo, long hWorkspace, int nWorkspaceOffset, long lWorkspaceSize, float fBeta, long hFilterDesc, long hWeightDiff, int nWeightOffset);


template <class T>
long Memory<T>::CreatePoolingDesc(long* phHandle)
{
	LONG lErr;
	hipdnnPoolingDescriptor_t desc = NULL;

	if (phHandle == NULL)
		return ERROR_PARAM_NULL;

	if (lErr = hipdnnCreatePoolingDescriptor(&desc))
		return lErr | ERROR_CUDNN_OFFSET;

	long hHandle = m_poolDesc.Allocate(desc);
	if (hHandle < 0)
	{
		hipdnnDestroyPoolingDescriptor(desc);
		return ERROR_MEMORY_OUT;
	}

	*phHandle = hHandle;
	return 0;
}

template long Memory<double>::CreatePoolingDesc(long* phHandle);
template long Memory<float>::CreatePoolingDesc(long* phHandle);


template <class T>
long Memory<T>::PoolingForward(long hHandle, long hPoolingDesc, T fAlpha, long hBottomDesc, long hBottomData, T fBeta, long hTopDesc, long hTopData)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnPoolingDescriptor_t pooldesc = GetPoolingDesc(hPoolingDesc);
	hipdnnTensorDescriptor_t topdesc = GetTensorDesc(hTopDesc);
	hipdnnTensorDescriptor_t btmdesc = GetTensorDesc(hBottomDesc);
	MemoryItem* pTopData;
	MemoryItem* pBtmData;

	if (lErr = m_memory.GetData(hTopData, &pTopData))
		return lErr;

	if (lErr = m_memory.GetData(hBottomData, &pBtmData))
		return lErr;

	T* topdata = (T*)pTopData->Data();
	T* btmdata = (T*)pBtmData->Data();

	if (lErr = hipdnnPoolingForward(cudnn, pooldesc, &fAlpha, btmdesc, btmdata, &fBeta, topdesc, topdata))
		return lErr | ERROR_CUDNN_OFFSET;

	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::PoolingForward(long hHandle, long hPoolingDesc, double dfAlpha, long hBottomDesc, long hBottomData, double dfBeta, long hTopDesc, long hTopData);
template long Memory<float>::PoolingForward(long hHandle, long hPoolingDesc, float fAlpha, long hBottomDesc, long hBottomData, float fBeta, long hTopDesc, long hTopData);


template <class T>
long Memory<T>::PoolingBackward(long hHandle, long hPoolingDesc, T fAlpha, long hTopDataDesc, long hTopData, long hTopDiffDesc, long hTopDiff, long hBottomDataDesc, long hBottomData, T fBeta, long hBottomDiffDesc, long hBottomDiff)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnPoolingDescriptor_t pooldesc = GetPoolingDesc(hPoolingDesc);
	hipdnnTensorDescriptor_t topdatadesc = GetTensorDesc(hTopDataDesc);
	hipdnnTensorDescriptor_t btmdatadesc = GetTensorDesc(hBottomDataDesc);
	hipdnnTensorDescriptor_t topdiffdesc = (hTopDataDesc == hTopDiffDesc) ? topdatadesc : GetTensorDesc(hTopDiffDesc);
	hipdnnTensorDescriptor_t btmdiffdesc = (hBottomDataDesc == hBottomDiffDesc) ? btmdatadesc : GetTensorDesc(hBottomDiffDesc);
	MemoryItem* pTopData;
	MemoryItem* pBtmData;
	MemoryItem* pTopDiff;
	MemoryItem* pBtmDiff;

	if (lErr = m_memory.GetData(hTopData, &pTopData))
		return lErr;

	if (lErr = m_memory.GetData(hBottomData, &pBtmData))
		return lErr;

	if (lErr = m_memory.GetData(hTopDiff, &pTopDiff))
		return lErr;

	if (lErr = m_memory.GetData(hBottomDiff, &pBtmDiff))
		return lErr;

	T* topdata = (T*)pTopData->Data();
	T* btmdata = (T*)pBtmData->Data();
	T* topdiff = (T*)pTopDiff->Data();
	T* btmdiff = (T*)pBtmDiff->Data();

	if (lErr = hipdnnPoolingBackward(cudnn, pooldesc, &fAlpha, topdatadesc, topdata, topdiffdesc, topdiff, btmdatadesc, btmdata, &fBeta, btmdiffdesc, btmdiff))
		return lErr | ERROR_CUDNN_OFFSET;

	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::PoolingBackward(long hHandle, long hPoolingDesc, double dfAlpha, long hTopDataDesc, long hTopData, long hTopDiffDesc, long hTopDiff, long hBottomDataDesc, long hBottomData, double dfBeta, long hBottomDiffDesc, long hBottomDiff);
template long Memory<float>::PoolingBackward(long hHandle, long hPoolingDesc, float fAlpha, long hTopDataDesc, long hTopData, long hTopDiffDesc, long hTopDiff, long hBottomDataDesc, long hBottomData, float fBeta, long hBottomDiffDesc, long hBottomDiff);


template <class T>
long Memory<T>::CreateDropoutDesc(long* phHandle)
{
	LONG lErr;
	hipdnnDropoutDescriptor_t desc = NULL;

	if (phHandle == NULL)
		return ERROR_PARAM_NULL;

	if (lErr = hipdnnCreateDropoutDescriptor(&desc))
		return lErr | ERROR_CUDNN_OFFSET;

	long hHandle = m_dropoutDesc.Allocate(desc);
	if (hHandle < 0)
	{
		hipdnnDestroyDropoutDescriptor(desc);
		return ERROR_MEMORY_OUT;
	}

	*phHandle = hHandle;
	return 0;
}

template long Memory<double>::CreateDropoutDesc(long* phHandle);
template long Memory<float>::CreateDropoutDesc(long* phHandle);


template <class T>
long Memory<T>::SetDropoutDesc(long hHandle, long hDropoutDesc, T fDropout, long hStates, long lSeed)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnDropoutDescriptor_t desc = GetDropoutDesc(hDropoutDesc);
	MemoryItem* pStates;
	
	if (lErr = m_memory.GetData(hStates, &pStates))
		return lErr;

	T* states = (T*)pStates->Data();
	size_t szStates = (size_t)pStates->Size();

	if (lErr = hipdnnSetDropoutDescriptor(desc, cudnn, (float)fDropout, states, szStates, (unsigned long long)lSeed))
		return lErr | ERROR_CUDNN_OFFSET;

	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::SetDropoutDesc(long hHandle, long hDropoutDesc, double fDropout, long hStates, long lSeed);
template long Memory<float>::SetDropoutDesc(long hHandle, long hDropoutDesc, float fDropout, long hStates, long lSeed);


template <class T>
long Memory<T>::GetDropoutInfo(long hHandle, long hBottomDesc, unsigned long* plState, unsigned long* plReserved)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnTensorDescriptor_t bottomDesc = GetTensorDesc(hBottomDesc);
	size_t szStates;
	size_t szReserved;

	if (plState == NULL || plReserved == NULL)
		return ERROR_PARAM_NULL;

	if (lErr = hipdnnDropoutGetStatesSize(cudnn, &szStates))
		return lErr | ERROR_CUDNN_OFFSET;

	if (lErr = cudnnDropoutGetReserveSpaceSize(bottomDesc, &szReserved))
		return lErr | ERROR_CUDNN_OFFSET;

	*plState = (unsigned long)szStates;
	*plReserved = (unsigned long)szReserved;

	return 0;
}

template long Memory<double>::GetDropoutInfo(long hHandle, long hBottomDesc, unsigned long* plState, unsigned long* plReserved);
template long Memory<float>::GetDropoutInfo(long hHandle, long hBottomDesc, unsigned long* plState, unsigned long* plReserved);


template <class T>
long Memory<T>::DropoutForward(long hHandle, long hDropoutDesc, long hBottomDesc, long hBottom, long hTopDesc, long hTop, long hReservedSpace)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnDropoutDescriptor_t desc = GetDropoutDesc(hDropoutDesc);
	hipdnnTensorDescriptor_t bottomDesc = GetTensorDesc(hBottomDesc);
	hipdnnTensorDescriptor_t topDesc = GetTensorDesc(hTopDesc);
	MemoryItem* pBottom;
	MemoryItem* pTop;
	MemoryItem* pReserved;

	if (lErr = m_memory.GetData(hBottom, &pBottom))
		return lErr;

	if (lErr = m_memory.GetData(hTop, &pTop))
		return lErr;

	if (lErr = m_memory.GetData(hReservedSpace, &pReserved))
		return lErr;

	T* bottom = (T*)pBottom->Data();
	T* top = (T*)pTop->Data();
	T* reserved = (T*)pReserved->Data();
	size_t szReserved = (size_t)pReserved->Size();

	if (lErr = cudnnDropoutForward(cudnn, desc, bottomDesc, bottom, topDesc, top, reserved, szReserved))
		return lErr | ERROR_CUDNN_OFFSET;

	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::DropoutForward(long hHandle, long hDropoutDesc, long hBottomDesc, long hBottom, long hTopDesc, long hTop, long hReservedSpace);
template long Memory<float>::DropoutForward(long hHandle, long hDropoutDesc, long hBottomDesc, long hBottom, long hTopDesc, long hTop, long hReservedSpace);


template <class T>
long Memory<T>::DropoutBackward(long hHandle, long hDropoutDesc, long hTopDesc, long hTop, long hBottomDesc, long hBottom, long hReservedSpace)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnDropoutDescriptor_t desc = GetDropoutDesc(hDropoutDesc);
	hipdnnTensorDescriptor_t topDesc = GetTensorDesc(hTopDesc);
	hipdnnTensorDescriptor_t bottomDesc = GetTensorDesc(hBottomDesc);
	MemoryItem* pTop;
	MemoryItem* pBottom;
	MemoryItem* pReserved;

	if (lErr = m_memory.GetData(hTop, &pTop))
		return lErr;

	if (lErr = m_memory.GetData(hBottom, &pBottom))
		return lErr;

	if (lErr = m_memory.GetData(hReservedSpace, &pReserved))
		return lErr;

	T* top = (T*)pTop->Data();
	T* bottom = (T*)pBottom->Data();
	T* reserved = (T*)pReserved->Data();
	size_t szReserved = (size_t)pReserved->Size();

	if (lErr = cudnnDropoutBackward(cudnn, desc, topDesc, top, bottomDesc, bottom, reserved, szReserved))
		return lErr | ERROR_CUDNN_OFFSET;

	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::DropoutBackward(long hHandle, long hDropoutDesc, long hTopDesc, long hTop, long hBottomDesc, long hBottom, long hReservedSpace);
template long Memory<float>::DropoutBackward(long hHandle, long hDropoutDesc, long hTopDesc, long hTop, long hBottomDesc, long hBottom, long hReservedSpace);


template <class T>
long Memory<T>::CreateLRNDesc(long* phHandle)
{
	LONG lErr;
	hipdnnLRNDescriptor_t desc = NULL;

	if (phHandle == NULL)
		return ERROR_PARAM_NULL;

	if (lErr = hipdnnCreateLRNDescriptor(&desc))
		return lErr | ERROR_CUDNN_OFFSET;

	long hHandle = m_lrnDesc.Allocate(desc);
	if (hHandle < 0)
	{
		hipdnnDestroyLRNDescriptor(desc);
		return ERROR_MEMORY_OUT;
	}

	*phHandle = hHandle;
	return 0;
}

template long Memory<double>::CreateLRNDesc(long* phHandle);
template long Memory<float>::CreateLRNDesc(long* phHandle);


template <class T> 
long Memory<T>::LRNForwardCC(long hHandle, long hNormDesc, T fAlpha, long hBottomDataDesc, long hBottomData, T fBeta, long hTopDataDesc, long hTopData)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnLRNDescriptor_t normdesc = GetLRNDesc(hNormDesc);
	hipdnnTensorDescriptor_t topdatadesc = GetTensorDesc(hTopDataDesc);
	hipdnnTensorDescriptor_t btmdatadesc = GetTensorDesc(hBottomDataDesc);
	MemoryItem* pBottomData;
	MemoryItem* pTopData;

	if (lErr = m_memory.GetData(hTopData, &pTopData))
		return lErr;

	if (lErr = m_memory.GetData(hBottomData, &pBottomData))
		return lErr;

	T* topdata = (T*)pTopData->Data();
	T* btmdata = (T*)pBottomData->Data();

	if (lErr = hipdnnLRNCrossChannelForward(cudnn, normdesc, HIPDNN_LRN_CROSS_CHANNEL, &fAlpha, btmdatadesc, btmdata, &fBeta, topdatadesc, topdata))
		return lErr | ERROR_CUDNN_OFFSET;

	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::LRNForwardCC(long hHandle, long hNormDesc, double fAlpha, long hBottomDesc, long hBottomData, double fBeta, long hTopDesc, long hTopData);
template long Memory<float>::LRNForwardCC(long hHandle, long hNormDesc, float fAlpha, long hBottomDesc, long hBottomData, float fBeta, long hTopDesc, long hTopData);


template <class T> 
long Memory<T>::LRNBackwardCC(long hHandle, long hNormDesc, T fAlpha, long hTopDataDesc, long hTopData, long hTopDiffDesc, long hTopDiff, long hBottomDataDesc, long hBottomData, T fBeta, long hBottomDiffDesc, long hBottomDiff)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnLRNDescriptor_t normdesc = GetLRNDesc(hNormDesc);
	hipdnnTensorDescriptor_t topdatadesc = GetTensorDesc(hTopDataDesc);
	hipdnnTensorDescriptor_t btmdatadesc = GetTensorDesc(hBottomDataDesc);
	hipdnnTensorDescriptor_t topdiffdesc = (hTopDataDesc == hTopDiffDesc) ? topdatadesc : GetTensorDesc(hTopDiffDesc);
	hipdnnTensorDescriptor_t btmdiffdesc = (hBottomDataDesc == hBottomDiffDesc) ? btmdatadesc : GetTensorDesc(hBottomDiffDesc);
	MemoryItem* pTopData;
	MemoryItem* pBtmData;
	MemoryItem* pTopDiff;
	MemoryItem* pBtmDiff;

	if (lErr = m_memory.GetData(hTopData, &pTopData))
		return lErr;

	if (lErr = m_memory.GetData(hBottomData, &pBtmData))
		return lErr;

	if (lErr = m_memory.GetData(hTopDiff, &pTopDiff))
		return lErr;

	if (lErr = m_memory.GetData(hBottomDiff, &pBtmDiff))
		return lErr;

	T* topdata = (T*)pTopData->Data();
	T* btmdata = (T*)pBtmData->Data();
	T* topdiff = (T*)pTopDiff->Data();
	T* btmdiff = (T*)pBtmDiff->Data();

	if (lErr = hipdnnLRNCrossChannelBackward(cudnn, normdesc, HIPDNN_LRN_CROSS_CHANNEL, &fAlpha, topdatadesc, topdata, topdiffdesc, topdiff, btmdatadesc, btmdata, &fBeta, btmdiffdesc, btmdiff))
		return lErr | ERROR_CUDNN_OFFSET;

	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::LRNBackwardCC(long hHandle, long hNormDesc, double fAlpha, long hTopDataDesc, long hTopData, long hTopDiffDesc, long hTopDiff, long hBottomDataDesc, long hBottomDadta, double fBeta, long hBottomDiffDesc, long hBottomDiff);
template long Memory<float>::LRNBackwardCC(long hHandle, long hNormDesc, float fAlpha, long hTopDataDesc, long hTopData, long hTopDiffDesc, long hTopDiff, long hBottomDataDesc, long hBottomDadta, float fBeta, long hBottomDiffDesc, long hBottomDiff);


template <class T> 
long Memory<T>::LCNForwardCC(long hHandle, long hNormDesc, T fAlpha, long hBottomDataDesc, long hBottomData, long hTemp1, long hTemp2, T fBeta, long hTopDataDesc, long hTopData)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnLRNDescriptor_t normdesc = GetLRNDesc(hNormDesc);
	hipdnnTensorDescriptor_t topdatadesc = GetTensorDesc(hTopDataDesc);
	hipdnnTensorDescriptor_t btmdatadesc = GetTensorDesc(hBottomDataDesc);
	MemoryItem* pBottomData;
	MemoryItem* pTopData;
	MemoryItem* pTemp1;
	MemoryItem* pTemp2;

	if (lErr = m_memory.GetData(hTopData, &pTopData))
		return lErr;

	if (lErr = m_memory.GetData(hBottomData, &pBottomData))
		return lErr;

	if (lErr = m_memory.GetData(hTemp1, &pTemp1))
		return lErr;

	if (lErr = m_memory.GetData(hTemp2, &pTemp2))
		return lErr;

	T* topdata = (T*)pTopData->Data();
	T* btmdata = (T*)pBottomData->Data();
	T* temp1 = (T*)pTemp1->Data();
	T* temp2 = (T*)pTemp2->Data();

	if (lErr = cudnnDivisiveNormalizationForward(cudnn, normdesc, CUDNN_DIVNORM_PRECOMPUTED_MEANS, &fAlpha, btmdatadesc, btmdata, NULL, temp1, temp2, &fBeta, topdatadesc, topdata))
		return lErr | ERROR_CUDNN_OFFSET;

	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::LCNForwardCC(long hHandle, long hNormDesc, double fAlpha, long hBottomDesc, long hBottomData, long hTemp1, long hTemp2, double fBeta, long hTopDesc, long hTopData);
template long Memory<float>::LCNForwardCC(long hHandle, long hNormDesc, float fAlpha, long hBottomDesc, long hBottomData, long hTemp1, long hTemp2, float fBeta, long hTopDesc, long hTopData);


template <class T> 
long Memory<T>::LCNBackwardCC(long hHandle, long hNormDesc, T fAlpha, long hBottomDataDesc, long hBottomData, long hTopDiff, long hTemp1, long hTemp2, T fBeta, long hBottomDiffDesc, long hBottomDiff)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnLRNDescriptor_t normdesc = GetLRNDesc(hNormDesc);
	hipdnnTensorDescriptor_t btmdatadesc = GetTensorDesc(hBottomDataDesc);
	hipdnnTensorDescriptor_t btmdiffdesc = (hBottomDataDesc == hBottomDiffDesc) ? btmdatadesc : GetTensorDesc(hBottomDiffDesc);
	MemoryItem* pBtmData;
	MemoryItem* pTopDiff;
	MemoryItem* pBtmDiff;
	MemoryItem* pTemp1;
	MemoryItem* pTemp2;

	if (lErr = m_memory.GetData(hBottomData, &pBtmData))
		return lErr;

	if (lErr = m_memory.GetData(hTopDiff, &pTopDiff))
		return lErr;

	if (lErr = m_memory.GetData(hBottomDiff, &pBtmDiff))
		return lErr;

	if (lErr = m_memory.GetData(hTemp1, &pTemp1))
		return lErr;

	if (lErr = m_memory.GetData(hTemp2, &pTemp2))
		return lErr;

	T* btmdata = (T*)pBtmData->Data();
	T* topdiff = (T*)pTopDiff->Data();
	T* btmdiff = (T*)pBtmDiff->Data();
	T* temp1 = (T*)pTemp1->Data();
	T* temp2 = (T*)pTemp2->Data();

	if (lErr = cudnnDivisiveNormalizationBackward(cudnn, normdesc, CUDNN_DIVNORM_PRECOMPUTED_MEANS, &fAlpha, btmdatadesc, btmdata, NULL, topdiff, temp1, temp2, &fBeta, btmdiffdesc, btmdiff, NULL))
		return lErr | ERROR_CUDNN_OFFSET;

	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::LCNBackwardCC(long hHandle, long hNormDesc, double fAlpha, long hBottomDataDesc, long hBottomData, long hTopDiff, long hTemp1, long hTemp2, double fBeta, long hBottomDiffDesc, long hBottomDiff);
template long Memory<float>::LCNBackwardCC(long hHandle, long hNormDesc, float fAlpha, long hBottomDataDesc, long hBottomData, long hTopDiff, long hTemp1, long hTemp2, float fBeta, long hBottomDiffDesc, long hBottomDiff);




template <class T>
long Memory<T>::TanhForward(long hHandle, T fAlpha, long hBottomDesc, long hBottomData, T fBeta, long hTopDesc, long hTopData)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnTensorDescriptor_t topdesc = GetTensorDesc(hTopDesc);
	hipdnnTensorDescriptor_t btmdesc = GetTensorDesc(hBottomDesc);
	MemoryItem* pTopData;
	MemoryItem* pBtmData;

	if (lErr = m_memory.GetData(hTopData, &pTopData))
		return lErr;

	if (lErr = m_memory.GetData(hBottomData, &pBtmData))
		return lErr;

	T* topdata = (T*)pTopData->Data();
	T* btmdata = (T*)pBtmData->Data();

#ifdef CUDNN_5
	hipdnnActivationDescriptor_t desc = GetActivationDesc(m_hGlobalActivationTanh);
	if (lErr = hipdnnActivationForward(cudnn, desc, &fAlpha, btmdesc, btmdata, &fBeta, topdesc, topdata))
		return lErr | ERROR_CUDNN_OFFSET;
#else
	if (lErr = hipdnnActivationForward(cudnn, HIPDNN_ACTIVATION_TANH, &fAlpha, btmdesc, btmdata, &fBeta, topdesc, topdata))
		return lErr | ERROR_CUDNN_OFFSET;
#endif

	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::TanhForward(long hHandle, double dfAlpha, long hBottomDesc, long hBottomData, double dfBeta, long hTopDesc, long hTopData);
template long Memory<float>::TanhForward(long hHandle, float fAlpha, long hBottomDesc, long hBottomData, float fBeta, long hTopDesc, long hTopData);


template <class T>
long Memory<T>::TanhBackward(long hHandle, T fAlpha, long hTopDataDesc, long hTopData, long hTopDiffDesc, long hTopDiff, long hBottomDataDesc, long hBottomData, T fBeta, long hBottomDiffDesc, long hBottomDiff)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnTensorDescriptor_t topdatadesc = GetTensorDesc(hTopDataDesc);
	hipdnnTensorDescriptor_t btmdatadesc = GetTensorDesc(hBottomDataDesc);
	hipdnnTensorDescriptor_t topdiffdesc = (hTopDataDesc == hTopDiffDesc) ? topdatadesc : GetTensorDesc(hTopDiffDesc);
	hipdnnTensorDescriptor_t btmdiffdesc = (hBottomDataDesc == hBottomDiffDesc) ? btmdatadesc : GetTensorDesc(hBottomDiffDesc);
	MemoryItem* pTopData;
	MemoryItem* pBtmData;
	MemoryItem* pTopDiff;
	MemoryItem* pBtmDiff;

	if (lErr = m_memory.GetData(hTopData, &pTopData))
		return lErr;

	if (lErr = m_memory.GetData(hBottomData, &pBtmData))
		return lErr;

	if (lErr = m_memory.GetData(hTopDiff, &pTopDiff))
		return lErr;

	if (lErr = m_memory.GetData(hBottomDiff, &pBtmDiff))
		return lErr;

	T* topdata = (T*)pTopData->Data();
	T* btmdata = (T*)pBtmData->Data();
	T* topdiff = (T*)pTopDiff->Data();
	T* btmdiff = (T*)pBtmDiff->Data();

#ifdef CUDNN_5
	hipdnnActivationDescriptor_t desc = GetActivationDesc(m_hGlobalActivationTanh);
	if (lErr = hipdnnActivationBackward(cudnn, desc, &fAlpha, topdatadesc, topdata, topdiffdesc, topdiff, btmdatadesc, btmdata, &fBeta, btmdiffdesc, btmdiff))
		return lErr | ERROR_CUDNN_OFFSET;
#else
	if (lErr = hipdnnActivationBackward(cudnn, HIPDNN_ACTIVATION_TANH, &fAlpha, topdatadesc, topdata, topdiffdesc, topdiff, btmdatadesc, btmdata, &fBeta, btmdiffdesc, btmdiff))
		return lErr | ERROR_CUDNN_OFFSET;
#endif

	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::TanhBackward(long hHandle, double dfAlpha, long hTopDataDesc, long hTopData, long hTopDiffDesc, long hTopDiff, long hBottomDataDesc, long hBottomData, double dfBeta, long hBottomDiffDesc, long hBottomDiff);
template long Memory<float>::TanhBackward(long hHandle, float fAlpha, long hTopDataDesc, long hTopData, long hTopDiffDesc, long hTopDiff, long hBottomDataDesc, long hBottomData, float fBeta, long hBottomDiffDesc, long hBottomDiff);


template <class T>
long Memory<T>::SigmoidForward(long hHandle, T fAlpha, long hBottomDesc, long hBottomData, T fBeta, long hTopDesc, long hTopData)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnTensorDescriptor_t topdesc = GetTensorDesc(hTopDesc);
	hipdnnTensorDescriptor_t btmdesc = GetTensorDesc(hBottomDesc);
	MemoryItem* pTopData;
	MemoryItem* pBtmData;

	if (lErr = m_memory.GetData(hTopData, &pTopData))
		return lErr;

	if (lErr = m_memory.GetData(hBottomData, &pBtmData))
		return lErr;

	T* topdata = (T*)pTopData->Data();
	T* btmdata = (T*)pBtmData->Data();

#ifdef CUDNN_5
	hipdnnActivationDescriptor_t desc = GetActivationDesc(m_hGlobalActivationSigmoid);
	if (lErr = hipdnnActivationForward(cudnn, desc, &fAlpha, btmdesc, btmdata, &fBeta, topdesc, topdata))
		return lErr | ERROR_CUDNN_OFFSET;
#else
	if (lErr = hipdnnActivationForward(cudnn, HIPDNN_ACTIVATION_SIGMOID, &fAlpha, btmdesc, btmdata, &fBeta, topdesc, topdata))
		return lErr | ERROR_CUDNN_OFFSET;
#endif

	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::SigmoidForward(long hHandle, double dfAlpha, long hBottomDesc, long hBottomData, double dfBeta, long hTopDesc, long hTopData);
template long Memory<float>::SigmoidForward(long hHandle, float fAlpha, long hBottomDesc, long hBottomData, float fBeta, long hTopDesc, long hTopData);


template <class T>
long Memory<T>::SigmoidBackward(long hHandle, T fAlpha, long hTopDataDesc, long hTopData, long hTopDiffDesc, long hTopDiff, long hBottomDataDesc, long hBottomData, T fBeta, long hBottomDiffDesc, long hBottomDiff)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnTensorDescriptor_t topdatadesc = GetTensorDesc(hTopDataDesc);
	hipdnnTensorDescriptor_t btmdatadesc = GetTensorDesc(hBottomDataDesc);
	hipdnnTensorDescriptor_t topdiffdesc = (hTopDataDesc == hTopDiffDesc) ? topdatadesc : GetTensorDesc(hTopDiffDesc);
	hipdnnTensorDescriptor_t btmdiffdesc = (hBottomDataDesc == hBottomDiffDesc) ? btmdatadesc : GetTensorDesc(hBottomDiffDesc);
	MemoryItem* pTopData;
	MemoryItem* pBtmData;
	MemoryItem* pTopDiff;
	MemoryItem* pBtmDiff;

	if (lErr = m_memory.GetData(hTopData, &pTopData))
		return lErr;

	if (lErr = m_memory.GetData(hBottomData, &pBtmData))
		return lErr;

	if (lErr = m_memory.GetData(hTopDiff, &pTopDiff))
		return lErr;

	if (lErr = m_memory.GetData(hBottomDiff, &pBtmDiff))
		return lErr;

	T* topdata = (T*)pTopData->Data();
	T* btmdata = (T*)pBtmData->Data();
	T* topdiff = (T*)pTopDiff->Data();
	T* btmdiff = (T*)pBtmDiff->Data();

#ifdef CUDNN_5
	hipdnnActivationDescriptor_t desc = GetActivationDesc(m_hGlobalActivationSigmoid);
	if (lErr = hipdnnActivationBackward(cudnn, desc, &fAlpha, topdatadesc, topdata, topdiffdesc, topdiff, btmdatadesc, btmdata, &fBeta, btmdiffdesc, btmdiff))
		return lErr | ERROR_CUDNN_OFFSET;
#else
	if (lErr = hipdnnActivationBackward(cudnn, HIPDNN_ACTIVATION_SIGMOID, &fAlpha, topdatadesc, topdata, topdiffdesc, topdiff, btmdatadesc, btmdata, &fBeta, btmdiffdesc, btmdiff))
		return lErr | ERROR_CUDNN_OFFSET;
#endif

	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::SigmoidBackward(long hHandle, double dfAlpha, long hTopDataDesc, long hTopData, long hTopDiffDesc, long hTopDiff, long hBottomDataDesc, long hBottomData, double dfBeta, long hBottomDiffDesc, long hBottomDiff);
template long Memory<float>::SigmoidBackward(long hHandle, float fAlpha, long hTopDataDesc, long hTopData, long hTopDiffDesc, long hTopDiff, long hBottomDataDesc, long hBottomData, float fBeta, long hBottomDiffDesc, long hBottomDiff);


template <class T>
long Memory<T>::ReLUForward(long hHandle, T fAlpha, long hBottomDesc, long hBottomData, T fBeta, long hTopDesc, long hTopData)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnTensorDescriptor_t topdesc = GetTensorDesc(hTopDesc);
	hipdnnTensorDescriptor_t btmdesc = GetTensorDesc(hBottomDesc);
	MemoryItem* pTopData;
	MemoryItem* pBtmData;

	if (lErr = m_memory.GetData(hTopData, &pTopData))
		return lErr;

	if (lErr = m_memory.GetData(hBottomData, &pBtmData))
		return lErr;

	T* topdata = (T*)pTopData->Data();
	T* btmdata = (T*)pBtmData->Data();

#ifdef CUDNN_5
	hipdnnActivationDescriptor_t desc = GetActivationDesc(m_hGlobalActivationRelu);
	if (lErr = hipdnnActivationForward(cudnn, desc, &fAlpha, btmdesc, btmdata, &fBeta, topdesc, topdata))
		return lErr | ERROR_CUDNN_OFFSET;
#else
	if (lErr = hipdnnActivationForward(cudnn, HIPDNN_ACTIVATION_RELU, &fAlpha, btmdesc, btmdata, &fBeta, topdesc, topdata))
		return lErr | ERROR_CUDNN_OFFSET;
#endif

	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::ReLUForward(long hHandle, double dfAlpha, long hBottomDesc, long hBottomData, double dfBeta, long hTopDesc, long hTopData);
template long Memory<float>::ReLUForward(long hHandle, float fAlpha, long hBottomDesc, long hBottomData, float fBeta, long hTopDesc, long hTopData);


template <class T>
long Memory<T>::ReLUBackward(long hHandle, T fAlpha, long hTopDataDesc, long hTopData, long hTopDiffDesc, long hTopDiff, long hBottomDataDesc, long hBottomData, T fBeta, long hBottomDiffDesc, long hBottomDiff)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnTensorDescriptor_t topdatadesc = GetTensorDesc(hTopDataDesc);
	hipdnnTensorDescriptor_t btmdatadesc = GetTensorDesc(hBottomDataDesc);
	hipdnnTensorDescriptor_t topdiffdesc = (hTopDataDesc == hTopDiffDesc) ? topdatadesc : GetTensorDesc(hTopDiffDesc);
	hipdnnTensorDescriptor_t btmdiffdesc = (hBottomDataDesc == hBottomDiffDesc) ? btmdatadesc : GetTensorDesc(hBottomDiffDesc);
	MemoryItem* pTopData;
	MemoryItem* pBtmData;
	MemoryItem* pTopDiff;
	MemoryItem* pBtmDiff;

	if (lErr = m_memory.GetData(hTopData, &pTopData))
		return lErr;

	if (lErr = m_memory.GetData(hBottomData, &pBtmData))
		return lErr;

	if (lErr = m_memory.GetData(hTopDiff, &pTopDiff))
		return lErr;

	if (lErr = m_memory.GetData(hBottomDiff, &pBtmDiff))
		return lErr;

	T* topdata = (T*)pTopData->Data();
	T* btmdata = (T*)pBtmData->Data();
	T* topdiff = (T*)pTopDiff->Data();
	T* btmdiff = (T*)pBtmDiff->Data();

#ifdef CUDNN_5
	hipdnnActivationDescriptor_t desc = GetActivationDesc(m_hGlobalActivationRelu);
	if (lErr = hipdnnActivationBackward(cudnn, desc, &fAlpha, topdatadesc, topdata, topdiffdesc, topdiff, btmdatadesc, btmdata, &fBeta, btmdiffdesc, btmdiff))
		return lErr | ERROR_CUDNN_OFFSET;
#else
	if (lErr = hipdnnActivationBackward(cudnn, HIPDNN_ACTIVATION_RELU, &fAlpha, topdatadesc, topdata, topdiffdesc, topdiff, btmdatadesc, btmdata, &fBeta, btmdiffdesc, btmdiff))
		return lErr | ERROR_CUDNN_OFFSET;
#endif

	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::ReLUBackward(long hHandle, double dfAlpha, long hTopDataDesc, long hTopData, long hTopDiffDesc, long hTopDiff, long hBottomDataDesc, long hBottomData, double dfBeta, long hBottomDiffDesc, long hBottomDiff);
template long Memory<float>::ReLUBackward(long hHandle, float fAlpha, long hTopDataDesc, long hTopData, long hTopDiffDesc, long hTopDiff, long hBottomDataDesc, long hBottomData, float fBeta, long hBottomDiffDesc, long hBottomDiff);



template <class T>
long Memory<T>::SoftmaxForward(long hHandle, T fAlpha, long hBottomDesc, long hBottomData, T fBeta, long hTopDesc, long hTopData)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnTensorDescriptor_t topdesc = GetTensorDesc(hTopDesc);
	hipdnnTensorDescriptor_t btmdesc = GetTensorDesc(hBottomDesc);
	MemoryItem* pTopData;
	MemoryItem* pBtmData;

	if (lErr = m_memory.GetData(hTopData, &pTopData))
		return lErr;

	if (lErr = m_memory.GetData(hBottomData, &pBtmData))
		return lErr;

	T* topdata = (T*)pTopData->Data();
	T* btmdata = (T*)pBtmData->Data();

	if (lErr = hipdnnSoftmaxForward(cudnn, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, &fAlpha, btmdesc, btmdata, &fBeta, topdesc, topdata))
		return lErr | ERROR_CUDNN_OFFSET;

	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::SoftmaxForward(long hHandle, double dfAlpha, long hBottomDesc, long hBottomData, double dfBeta, long hTopDesc, long hTopData);
template long Memory<float>::SoftmaxForward(long hHandle, float fAlpha, long hBottomDesc, long hBottomData, float fBeta, long hTopDesc, long hTopData);


template <class T>
long Memory<T>::SoftmaxBackward(long hHandle, T fAlpha, long hTopDataDesc, long hTopData, long hTopDiffDesc, long hTopDiff, T fBeta, long hBottomDiffDesc, long hBottomDiff)
{
	LONG lErr;
	hipdnnHandle_t cudnn = GetCuDNN(hHandle);
	hipdnnTensorDescriptor_t topdatadesc = GetTensorDesc(hTopDataDesc);
	hipdnnTensorDescriptor_t topdiffdesc = (hTopDataDesc == hTopDiffDesc) ? topdatadesc : GetTensorDesc(hTopDiffDesc);
	hipdnnTensorDescriptor_t btmdiffdesc = GetTensorDesc(hBottomDiffDesc);
	MemoryItem* pTopData;
	MemoryItem* pTopDiff;
	MemoryItem* pBtmDiff;

	if (lErr = m_memory.GetData(hTopData, &pTopData))
		return lErr;

	if (lErr = m_memory.GetData(hTopDiff, &pTopDiff))
		return lErr;

	if (lErr = m_memory.GetData(hBottomDiff, &pBtmDiff))
		return lErr;

	T* topdata = (T*)pTopData->Data();
	T* topdiff = (T*)pTopDiff->Data();
	T* btmdiff = (T*)pBtmDiff->Data();

	if (lErr = hipdnnSoftmaxBackward(cudnn, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, &fAlpha, topdatadesc, topdata, topdiffdesc, topdiff, &fBeta, btmdiffdesc, btmdiff))
		return lErr | ERROR_CUDNN_OFFSET;

	return HIPDNN_STATUS_SUCCESS;
}

template long Memory<double>::SoftmaxBackward(long hHandle, double dfAlpha, long hTopDataDesc, long hTopData, long hTopDiffDesc, long hTopDiff, double dfBeta, long hBottomDiffDesc, long hBottomDiff);
template long Memory<float>::SoftmaxBackward(long hHandle, float fAlpha, long hTopDataDesc, long hTopData, long hTopDiffDesc, long hTopDiff, float fBeta, long hBottomDiffDesc, long hBottomDiff);




//end memory.cu